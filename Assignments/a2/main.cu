#include<iostream>
// #include<sys/time.h>
#include<hip/hip_runtime.h>
using namespace std;


// write kernels here...
__global__ void sum(int *A, int *B, int *X){
	unsigned int id1 = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int id2 = gridDim.x * threadIdx.x + blockIdx.x;
	X[id1] = A[id1] + B[id2];
}

__global__ void prod1(int *C, int *D, int *X, int r){
	extern __shared__ int s[];
	unsigned int id1 = r * blockIdx.x;
	unsigned int id2 = r * threadIdx.x;
	for(int i=0;i<r;i++){
		s[i] = C[id1 + i];
	}
	__syncthreads();
	int temp = 0;
	for(int i=0;i<r;i++){
		temp += s[i] * D[id2 + i];
	}
	X[blockIdx.x * blockDim.x + threadIdx.x] = temp;
}

__global__ void prod2(int *A, int *B, int *X, int q){
	extern __shared__ int s[];
	unsigned int id1 = q * blockIdx.x;
	unsigned int id2 = threadIdx.x;
	for(int i=0;i<q;i++){
		s[i] = A[id1 + i];
	}
	__syncthreads();
	int temp = 0;
	for(int i=0;i<q;i++){
		temp += s[i] * B[id2 + i * blockDim.x];
	}
	X[blockIdx.x * blockDim.x + threadIdx.x] = temp;
}

// function to compute the output matrix
void compute(int p, int q, int r, int s, int *h_matrixA, int *h_matrixB, 
	         int *h_matrixC, int *h_matrixD, int *h_matrixX) {
	// variable declarations...
	int *d_matrixA, *d_matrixB, *d_matrixC, *d_matrixD, *d_matrixX;
	int *temp1, *temp2;

	// allocate memory...
	hipMalloc(&d_matrixA, (p*q)*sizeof(int));
	hipMalloc(&d_matrixB, (q*p)*sizeof(int));
	hipMalloc(&d_matrixC, (q*r)*sizeof(int));
	hipMalloc(&d_matrixD, (s*r)*sizeof(int));
	hipMalloc(&d_matrixX, (p*s)*sizeof(int));

	hipMalloc(&temp1, (p*q)*sizeof(int));
	hipMalloc(&temp2, (q*s)*sizeof(int));

	// copy the values...
	hipMemcpy(d_matrixA, h_matrixA, (p*q)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_matrixB, h_matrixB, (q*p)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_matrixC, h_matrixC, (q*r)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_matrixD, h_matrixD, (s*r)*sizeof(int), hipMemcpyHostToDevice);
	
	// call the kernels for doing required computations...
	sum<<<p,q>>>(d_matrixA, d_matrixB, temp1);
	prod1<<<q,s,r*sizeof(int)>>>(d_matrixC, d_matrixD, temp2, r);
	prod2<<<p,s,q*sizeof(int)>>>(temp1,temp2,d_matrixX, q);

	// copy the result back...
	hipMemcpy(h_matrixX, d_matrixX, (p*s)*sizeof(int), hipMemcpyDeviceToHost);
	
	// deallocate the memory...
	hipFree(d_matrixA);
	hipFree(d_matrixB);
	hipFree(d_matrixC);
	hipFree(d_matrixD);
	hipFree(d_matrixX);

	hipFree(temp1);
	hipFree(temp2);
}

// function to read the input matrices from the input file
void readMatrix(FILE *inputFilePtr, int *matrix, int rows, int cols) {
	for(int i=0; i<rows; i++) {
		for(int j=0; j<cols; j++) {
			fscanf(inputFilePtr, "%d", &matrix[i*cols+j]);
		}
	}
}

// function to write the output matrix into the output file
void writeMatrix(FILE *outputFilePtr, int *matrix, int rows, int cols) {
	for(int i=0; i<rows; i++) {
		for(int j=0; j<cols; j++) {
			fprintf(outputFilePtr, "%d ", matrix[i*cols+j]);
		}
		fprintf(outputFilePtr, "\n");
	}
}

int main(int argc, char **argv) {
	// variable declarations
	int p, q, r, s;
	int *matrixA, *matrixB, *matrixC, *matrixD, *matrixX;
	// struct timeval t1, t2;
	double seconds, microSeconds;

	// get file names from command line
	char *inputFileName = argv[1];
	char *outputFileName = argv[2];

	// file pointers
	FILE *inputFilePtr, *outputFilePtr;
    
    inputFilePtr = fopen(inputFileName, "r");
	if(inputFilePtr == NULL) {
	    printf("Failed to open the input file.!!\n"); 
		return 0;
	}

	// read input values
	fscanf(inputFilePtr, "%d %d %d %d", &p, &q, &r, &s);

	// allocate memory and read input matrices
	matrixA = (int*) malloc(p * q * sizeof(int));
	matrixB = (int*) malloc(q * p * sizeof(int));
	matrixC = (int*) malloc(q * r * sizeof(int));
	matrixD = (int*) malloc(s * r * sizeof(int));
	readMatrix(inputFilePtr, matrixA, p, q);
	readMatrix(inputFilePtr, matrixB, q, p);
	readMatrix(inputFilePtr, matrixC, q, r);
	readMatrix(inputFilePtr, matrixD, s, r);

	// allocate memory for output matrix
	matrixX = (int*) malloc(p * s * sizeof(int));

	// call compute function to get the output matrix. it is expected that 
	// the compute function will store the result in matrixX.
	// gettimeofday(&t1, NULL);
	compute(p, q, r, s, matrixA, matrixB, matrixC, matrixD, matrixX);
	hipDeviceSynchronize();
	// gettimeofday(&t2, NULL);

	// print the time taken by the compute function
	// seconds = t2.tv_sec - t1.tv_sec;
	// microSeconds = t2.tv_usec - t1.tv_usec;
	// printf("Time taken (ms): %.3f\n", 1000*seconds + microSeconds/1000);

	// store the result into the output file
	outputFilePtr = fopen(outputFileName, "w");
	writeMatrix(outputFilePtr, matrixX, p, s);

	// close files
    fclose(inputFilePtr);
    fclose(outputFilePtr);

	// deallocate memory
	free(matrixA);
	free(matrixB);
	free(matrixC);
	free(matrixD);
	free(matrixX);

	return 0;
}