#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

__global__ void schedule(int n, unsigned int *id, int *executionTime, int *priority, int *result, int *priorityMap, int *coreEndTime, int *coreCurrId){
    __shared__ int currTime;
    __shared__ int lockVar[1];
    __shared__ int flag;
    unsigned int tid = threadIdx.x;
    int core = -1;
    int old = 0;
    currTime = 0;
    lockVar[0] = 0;
    flag = 0;
    while(id[0] < n){
        do{
            old = atomicCAS(lockVar, 0, 1);
            if(old == 0){
                core = priorityMap[priority[id[0]]];
                if(core == -1){
                    int minCore = 1001;
                    for(int i=0;i<blockDim.x;i++){
                        if(coreEndTime[i] <= currTime){
                            minCore = (minCore < i) ? minCore : i;
                        }
                    }
                    priorityMap[priority[id[0]]] = minCore;
                    core = minCore;
                }
                if(coreCurrId[core] == -1){
                    coreCurrId[core] = id[0];
                    if(currTime < coreEndTime[core]){
                        currTime = coreEndTime[core];
                    }else{
                        coreEndTime[core] = currTime;
                    }
                    id[0] += 1;
                    printf("%d :: %d\n",id[0], currTime);
                }else{
                    flag = 1;
                }

                lockVar[0] = 0;
                old = 0;
            }
        } while(old != 0 && flag != 1);

        int tempId = coreCurrId[tid];
        if(tempId != -1){
            coreEndTime[tid] += executionTime[tempId];
            result[tempId] = coreEndTime[tid];
            coreCurrId[tid] = -1;
        }
    
        __syncthreads();
        flag = 0;
    }
}

//Complete the following function
void operations ( int m, int n, int *executionTime, int *priority, int *result )  {
    int *dExecutionTime, *dPriority, *dResult, *priorityMap, *coreEndTime, *coreCurrId;
    unsigned int *id;

    hipMalloc(&dExecutionTime, n*sizeof(int));
    hipMalloc(&dPriority, n*sizeof(int));
    hipMalloc(&dResult, n*sizeof(int));
    hipMalloc(&priorityMap, m*sizeof(int));
    hipMalloc(&coreEndTime, m*sizeof(int));
    hipMalloc(&coreCurrId, m*sizeof(int));
    hipMalloc(&id, sizeof(unsigned int));

    hipMemcpy(dExecutionTime, executionTime, n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dPriority, priority, n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dResult, result, n*sizeof(int), hipMemcpyHostToDevice);

    hipMemset(priorityMap, -1, m*sizeof(int));
    hipMemset(coreEndTime, 0, m*sizeof(int));
    hipMemset(coreCurrId, -1, m*sizeof(int));

    schedule<<<1,m>>>(n, id, dExecutionTime, dPriority, dResult, priorityMap, coreEndTime, coreCurrId);

    hipMemcpy(result, dResult, n*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dExecutionTime);
    hipFree(dPriority);
    hipFree(dResult);
    hipFree(priorityMap);
    hipFree(coreEndTime);
    hipFree(id);
}

int main(int argc,char **argv)
{
    int m,n;
    //Input file pointer declaration
    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");
    
    //Checking if file ptr is NULL
    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &m );      //scaning for number of cores
    fscanf( inputfilepointer, "%d", &n );      //scaning for number of tasks
   
   //Taking execution time and priorities as input	
    int *executionTime = (int *) malloc ( n * sizeof (int) );
    int *priority = (int *) malloc ( n * sizeof (int) );
    for ( int i=0; i< n; i++ )  {
            fscanf( inputfilepointer, "%d", &executionTime[i] );
    }

    for ( int i=0; i< n; i++ )  {
            fscanf( inputfilepointer, "%d", &priority[i] );
    }

    //Allocate memory for final result output 
    int *result = (int *) malloc ( (n) * sizeof (int) );
    for ( int i=0; i<n; i++ )  {
        result[i] = 0;
    }
    
     hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start,0);

    //==========================================================================================================
	

	operations ( m, n, executionTime, priority, result ); 
	
    //===========================================================================================================
    
    
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken by function to execute is: %.6f ms\n", milliseconds);
    
    // Output file pointer declaration
    char *outputfilename = argv[2]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    //Total time of each task: Final Result
    for ( int i=0; i<n; i++ )  {
        fprintf( outputfilepointer, "%d ", result[i]);
    }

    fclose( outputfilepointer );
    fclose( inputfilepointer );
    
    std::free(executionTime);
    std::free(priority);
    std::free(result);
    
    
    
}
